#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>


hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

#define EXPCNT 100000
#define N 5
#define DENOM 623360743125120
#define KNS 256
#define WARP 32


__global__ void setup_kernel(hiprandState* state) {

    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    hiprand_init(idx, idx, 0, &state[idx]);
}

__device__ void swap(int* arr,int i, int j) {
    int tmp = arr[i];
    arr[i] = arr[j];
    arr[j] = tmp;
}

__device__ int analysis(int* arr) {
    int stack[5][5] = { 0 };
    int maxi[5] = { -1,-1,-1,-1,-1 };
    for (int i = 0; i < 5; i++) {
        for (int idx = i * 5; idx < i * 5 + 5; idx++) {
            stack[arr[idx] / 5][i] += 1;
        }
    }
  
    int mi = -1;
    for (int i = 0; i < 5; i++) {
        int twoz = 0;
        bool broke = false;
        for (int j = 0; j < 5; j++) {
            int v = stack[i][j];
            if (v > 2) {
                if (maxi[j] == -1) {
                    maxi[j] = 1;
                    broke = true;
                    break;
                }
                else {
                    return 0;
                }
            }
            else {
                if (v == 2) {
                    //printf("TWO\n");
                    if (twoz == 1) {
                        return 0;
                    }
                    else {
                        mi = j;
                        twoz += 1;
                    }
                }
            }
        }
        if (!broke) {
            if (twoz == 1) {
                if (maxi[mi] == -1) {
                    maxi[mi] = 1;
                }
                else {
                    return 0;
                }
            }
            else {
                return 0;
            }
            
        }
    }
    return 1;
}

__global__ void expKernel(long long *acc, hiprandState* my_curandstate) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;

    for (int i = 0; i < EXPCNT; i++) {
        int conf[25] = { 0,1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16,17,18,19,20,21,22,23,24 };
        for (int js = 0; js < 25; js++) {
            float myrandf = hiprand_uniform(&(my_curandstate[idx]));
            int myrand = ceilf(myrandf * 24);
            //printf("%d\n", myrand);
            swap(conf, js, myrand);
        }
        int a = analysis(conf);
        // printf("%d\n", a);
        acc[idx] += a;
    }
    
    /*printf("idx:%d: %d\n", idx,myrand);*/
}

int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };
    long long acc[KNS*WARP] = { 0 };
    long long* acc_d;
    hipMalloc(&acc_d, sizeof(long long) * KNS * WARP);
    hipMemcpy(acc_d, acc, sizeof(long long) * KNS * WARP, hipMemcpyHostToDevice);

    hiprandState* d_state;
    hipMalloc(&d_state, sizeof(hiprandState)*KNS);
   
    setup_kernel << <WARP, KNS >> > (d_state);
    expKernel <<< WARP, KNS >>> (acc_d,d_state);
    hipDeviceSynchronize();
    hipMemcpy(acc, acc_d, sizeof(long long) * KNS * WARP, hipMemcpyDeviceToHost);
    long long summ = 0;
    for (auto i = 0; i < KNS * WARP; i++) {
        // printf("%ld | ", acc[i]);
        summ += acc[i];
    }
    printf("%ld : %ld\n", summ, WARP*KNS*EXPCNT);
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    auto cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}


